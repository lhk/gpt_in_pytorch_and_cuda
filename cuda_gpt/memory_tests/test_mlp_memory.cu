#include "../modules.cu"
#include <iostream>
#include <memory>
#include <stdexcept>

size_t get_gpu_memory_used() {
    size_t free_bytes, total_bytes;
    hipError_t err = hipMemGetInfo(&free_bytes, &total_bytes);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to get GPU memory info: " + std::string(hipGetErrorString(err)));
    }
    return total_bytes - free_bytes;
}

int main() {
    std::cout << "=== MLP Memory Test ===" << std::endl;
    
    try {
        // Initialize CUDA context and get baseline memory
        hipSetDevice(0);
        size_t initial_memory = get_gpu_memory_used();
        std::cout << "Initial GPU memory: " << (initial_memory / 1024) << " KB" << std::endl;
        
        // Setup dummy weights
        int d_model = 64, d_ff = 128, max_seq_len = 32;
        
        std::vector<float> w1(d_model * d_ff, 0.01f);
        std::vector<float> b1(d_ff, 0.0f);
        std::vector<float> w2(d_ff * d_model, 0.01f);
        std::vector<float> b2(d_model, 0.0f);
        
        // Test 1: Manual allocation/deallocation
        std::cout << "Testing manual allocation..." << std::endl;
        MLP* mlp_manual = new MLP(
            d_model, d_ff, max_seq_len,
            w1.data(), b1.data(), w2.data(), b2.data()
        );
        delete mlp_manual;
        
        // Test 2: Smart pointer (RAII)
        std::cout << "Testing smart pointer allocation..." << std::endl;
        {
            auto mlp_smart = std::make_unique<MLP>(
                d_model, d_ff, max_seq_len,
                w1.data(), b1.data(), w2.data(), b2.data()
            );
        }
        
        // Test 3: Multiple allocations
        std::cout << "Testing multiple allocations..." << std::endl;
        std::vector<std::unique_ptr<MLP>> mlp_instances;
        for (int i = 0; i < 5; i++) {
            mlp_instances.push_back(std::make_unique<MLP>(
                d_model, d_ff, max_seq_len,
                w1.data(), b1.data(), w2.data(), b2.data()
            ));
        }
        mlp_instances.clear();
        
        // Verify memory has returned to baseline
        size_t final_memory = get_gpu_memory_used();
        std::cout << "Final GPU memory: " << (final_memory / 1024) << " KB" << std::endl;
        
        if (final_memory != initial_memory) {
            throw std::runtime_error("MEMORY LEAK DETECTED: Initial=" + 
                std::to_string(initial_memory / 1024) + "KB, Final=" + 
                std::to_string(final_memory / 1024) + "KB, Difference=" + 
                std::to_string((long long)final_memory - (long long)initial_memory) + " bytes");
        }
        
        std::cout << "MLP memory test PASSED - no leaks detected" << std::endl;
        
    } catch (const std::exception& e) {
        std::cerr << "ERROR: " << e.what() << std::endl;
        return 1;
    }
    
    return 0;
}
